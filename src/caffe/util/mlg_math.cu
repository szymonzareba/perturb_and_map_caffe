#include "hip/hip_runtime.h"
#include "caffe/common.hpp"
#include "caffe/util/mlg_math.hpp"

namespace caffe {

template <typename Dtype>
__global__ void binarization_kernel(const int count, const Dtype threshold, const Dtype* x, Dtype* y) {
  CUDA_KERNEL_LOOP(index, count) {
   if(x[index] > threshold){
	y[index] = (Dtype) 1.;
   }else{
	y[index] = (Dtype) 0.;
   }
  }
}

template <typename Dtype>
__global__ void replicate_kernel(const int xcount, const int repxcount, const Dtype* x, Dtype* repx) {
  CUDA_KERNEL_LOOP(index, repxcount) {
   repx[index] = x[index % xcount];
  }
}

template <typename Dtype>
__global__ void sample_ge0_kernel(const int n, Dtype* y) {
  CUDA_KERNEL_LOOP(index, n) {
	if(y[index] > (Dtype)0.){
		y[index] = (Dtype) 1.;
	}else{
		y[index] = (Dtype) 0.;
	}
  }
}

template <typename Dtype>
__global__ void sample_ge0_kernel(const int n, const Dtype* src, Dtype* dst) {
  CUDA_KERNEL_LOOP(index, n) {
	if(src[index] > (Dtype)0.){
		dst[index] = (Dtype) 1.;
	}else{
		dst[index] = (Dtype) 0.;
	}
  }
}

template <typename Dtype>
__global__ void sample_ge0_5_kernel(const int n, Dtype* y) {
  CUDA_KERNEL_LOOP(index, n) {
	if(y[index] > (Dtype)0.5){
		y[index] = (Dtype) 1.;
	}else{
		y[index] = (Dtype) 0.;
	}
  }
}

template <typename Dtype>
__global__ void add_scaled_kernel(const int n, const Dtype alpha, const Dtype* a, const Dtype beta, const Dtype* b, Dtype* x) {
  CUDA_KERNEL_LOOP(index, n) {
	x[index] = alpha * a[index] + beta * b[index];
  }
}

template <typename Dtype>
__global__ void relax_0_1_kernel(const int n, Dtype* x) {
  CUDA_KERNEL_LOOP(index, n) {
	if(x[index] > (Dtype)1.){
		x[index] = (Dtype)1.;
	}

	if(x[index] < (Dtype)0.){
		x[index] = (Dtype)0.;
	}
  }
}

template <typename Dtype>
__global__ void negate_0_1_kernel(const int n, Dtype* x){
	CUDA_KERNEL_LOOP(index, n){
		x[index] = (Dtype)1. - x[index];
	}
}

template <typename Dtype>
__global__ void add_with_mask_kernel(const int n, const Dtype* a, const Dtype* bMask, const Dtype* b, Dtype* x){
	CUDA_KERNEL_LOOP(index, n){
		x[index] = a[index] + bMask[index] * b[index];
	}
}

template <typename Dtype>
__global__ void negate_g_kernel(const int n, const Dtype threshold, const Dtype* mask, Dtype* x){
	CUDA_KERNEL_LOOP(index, n){
		if(mask[index] > threshold){
			x[index] = - x[index];
		}
	}
}

template <typename Dtype>
__global__ void negate_l_kernel(const int n, const Dtype threshold, const Dtype* mask, Dtype* x){
	CUDA_KERNEL_LOOP(index, n){
		if(mask[index] < threshold){
			x[index] = - x[index];
		}
	}
}

template <typename Dtype>
__global__ void negate_0_1_g_kernel(const int n, const Dtype threshold, const Dtype* mask, Dtype* x){
	CUDA_KERNEL_LOOP(index, n){
		if(mask[index] > threshold){
			x[index] = (Dtype)1. - x[index];
		}
	}
}

template <typename Dtype>
__global__ void negate_0_1_l_kernel(const int n, const Dtype threshold, const Dtype* mask, Dtype* x){
	CUDA_KERNEL_LOOP(index, n){
		if(mask[index] < threshold){
			x[index] = (Dtype)1. - x[index];
		}
	}
}

template
__global__ void binarization_kernel(const int count, const float threshold, const float* x, float* y);

template
__global__ void binarization_kernel(const int count, const double threshold, const double* x, double* y);

template
__global__ void replicate_kernel<float>(const int xcount, const int repxcount, const float* x, float* repx);

template
__global__ void replicate_kernel<double>(const int xcount, const int repxcount, const double* x, double* repx);

template
__global__ void sample_ge0_kernel<float>(const int n, float* y);

template
__global__ void sample_ge0_kernel<double>(const int n, double* y);

template
__global__ void sample_ge0_kernel<float>(const int n, const float* src, float* dst);

template
__global__ void sample_ge0_kernel<double>(const int n, const double* src, double* dst);

template
__global__ void sample_ge0_5_kernel<float>(const int n, float* y);

template
__global__ void sample_ge0_5_kernel<double>(const int n, double* y);

template
__global__ void add_scaled_kernel<float>(const int n, const float alpha, const float* a, const float beta, const float* b, float* x);

template
__global__ void add_scaled_kernel<double>(const int n, const double alpha, const double* a, const double beta, const double* b, double* x);

template
__global__ void relax_0_1_kernel<float>(const int n, float* x);

template
__global__ void relax_0_1_kernel<double>(const int n, double* x);

template
__global__ void negate_0_1_kernel<float>(const int n, float* x);

template
__global__ void negate_0_1_kernel<double>(const int n, double* x);

template
__global__ void negate_g_kernel<float>(const int n, const float threshold, const float* mask, float* x);

template
__global__ void negate_g_kernel<double>(const int n, const double threshold, const double* mask, double* x);

template
__global__ void negate_l_kernel<float>(const int n, const float threshold, const float* mask, float* x);

template
__global__ void negate_l_kernel<double>(const int n, const double threshold, const double* mask, double* x);

template
__global__ void negate_0_1_g_kernel<float>(const int n, const float threshold, const float* mask, float* x);

template
__global__ void negate_0_1_g_kernel<double>(const int n, const double threshold, const double* mask, double* x);

template
__global__ void negate_0_1_l_kernel<float>(const int n, const float threshold, const float* mask, float* x);

template
__global__ void negate_0_1_l_kernel<double>(const int n, const double threshold, const double* mask, double* x);

template
__global__ void add_with_mask_kernel<float>(const int n, const float* a, const float* bMask, const float* b, float* x);

template
__global__ void add_with_mask_kernel<double>(const int n, const double* a, const double* bMask, const double* b, double* x);
}
