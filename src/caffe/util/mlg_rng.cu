#include "hip/hip_runtime.h"
#include "caffe/util/mlg_rng.hpp"


namespace caffe {

__global__ void mlg_init_rng_kernel(unsigned int seed, int n, hiprandState_t* states) {
	CUDA_KERNEL_LOOP(index, n) {
  /* we have to initialize the state */
  hiprand_init(seed, /* the seed can be the same for each core, here we pass the time in from the CPU */
		  	  index, /* the sequence number should be different for each core (unless you want all
                             cores to get the same sequence of numbers for some reason - use thread id! */
              0, /* the offset is how much extra we advance in the sequence for each call, can be 0 */
              &states[index]);
	}
}

template <typename Dtype>
__global__ void mlg_uniform_kernel(hiprandState_t* states, int n, Dtype* numbers) {
 CUDA_KERNEL_LOOP(index, n) {
  numbers[index] = hiprand_uniform(&states[index]);
 }
}

template <typename Dtype>
__global__ void mlg_gumbel_kernel(int n, Dtype* numbers) {
 CUDA_KERNEL_LOOP(index, n) {
  numbers[index] = log( numbers[index] ) - log( (Dtype)1.0 - numbers[index] );
 }
}

template <typename Dtype>
void MLGRNG<Dtype>::mlg_gpu_uniform(const int N, Dtype* data){
	if(stateCount < N)
	{
		//LOG(INFO) << "Random states reshaped " << stateCount << " to " << N << " " << std::endl;
		if(stateCount != 0)
		{
			hipFree(states);
		}
		hipMalloc((void**) &states, N * sizeof(hiprandState_t));
		stateCount = N;

		mlg_init_rng_kernel<<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(time(0), N, states);
		CUDA_POST_KERNEL_CHECK;

		//LOG(INFO) << "Done" << std::endl;
	}

	mlg_uniform_kernel<<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(states, N, data);
	CUDA_POST_KERNEL_CHECK;

}

template <typename Dtype>
__global__ void mlg_fix_0_1_kernel(int n, Dtype* numbers) {
 CUDA_KERNEL_LOOP(index, n) {
  if(numbers[index] < MLG_MIN_UNI){
   numbers[index] = MLG_MIN_UNI;
  }
  if(numbers[index] > MLG_MAX_UNI){
   numbers[index] = MLG_MAX_UNI;
  }
 }
}

template <typename Dtype>
void MLGRNG<Dtype>::mlg_gpu_gumbel(const int N, Dtype* data){
	mlg_gpu_uniform(N, data);

	// add or substract small value to avoid nans etc
	mlg_fix_0_1_kernel<<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(N, data);
	CUDA_POST_KERNEL_CHECK;

	mlg_gumbel_kernel<<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(N, data);
	CUDA_POST_KERNEL_CHECK;
}




template <typename Dtype>
__global__ void mlg_set_index_kernel(int n, int* numbers) {
 CUDA_KERNEL_LOOP(index, n) {
  numbers[index] = index;
 }
}


template <typename Dtype>
void MLGRNG<Dtype>::mlg_gpu_permutation(const int N, int* data){

	int* tmp = new int[N];
	mlg_cpu_permutation(N, tmp);
	hipMemcpy(data, tmp, N * sizeof(int), hipMemcpyHostToDevice);
	delete tmp;

	/*
	mlg_set_index_kernel<Dtype><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(N, data);
	CUDA_POST_KERNEL_CHECK;

	int* range;
	hipMalloc((void**) &range, N * sizeof(int));

	mlg_gpu_range(N, 0, N-1, range);

	int* local2 = new int[N];
	hipMemcpy(local2, range, N * sizeof(int), hipMemcpyDeviceToHost);

	int* tmp;
	hipMalloc((void**) &tmp,  sizeof(int));
	for(int i = 0; i < N; i++){
		hipMemcpy(tmp, data + i, 1 * sizeof(int), hipMemcpyDeviceToDevice);
		hipMemcpy(data + i, data + (range[i]), 1 * sizeof(int), hipMemcpyDeviceToDevice);
		hipMemcpy(data + (range[i]), tmp, 1 * sizeof(int), hipMemcpyDeviceToDevice);
	}

	int* local1 = new int[N];
	hipMemcpy(local1, data, N * sizeof(int), hipMemcpyDeviceToHost);

	hipFree(range);
	*/
}

template <typename Dtype>
__global__ void mlg_set_min_max_kernel(int n, int min, int max, Dtype* rand, int* numbers) {
 CUDA_KERNEL_LOOP(index, n) {
  numbers[index] = (((int)(rand[index] * 100000)) % (max - min + 1) ) + min;
 }
}

template <typename Dtype>
void MLGRNG<Dtype>::mlg_gpu_range(const int N, const int min, const int max, int* data){
	Dtype* tmp;
	hipMalloc((void**) &tmp, N * sizeof(Dtype));
	mlg_gpu_uniform(N, tmp);

	mlg_set_min_max_kernel<Dtype><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(N, min, max, tmp, data);
	CUDA_POST_KERNEL_CHECK;

	hipFree(tmp);
}

template
void MLGRNG<float>::mlg_gpu_uniform( \
		const int N, \
		float* data);

template
void MLGRNG<double>::mlg_gpu_uniform( \
		const int N, \
		double* data);

template
void MLGRNG<float>::mlg_gpu_gumbel( \
		const int N, \
		float* data);

template
void MLGRNG<double>::mlg_gpu_gumbel( \
		const int N, \
		double* data);

template
void MLGRNG<float>::mlg_gpu_permutation(const int N, int* data);

template
void MLGRNG<double>::mlg_gpu_permutation(const int N, int* data);

template
void MLGRNG<float>::mlg_gpu_range(const int N, const int min, const int max, int* data);

template
void MLGRNG<double>::mlg_gpu_range(const int N, const int min, const int max, int* data);
}
